#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#ifdef USE_CUBLAS
#include <cublas_v2.h>
#endif
#include <cmath>

using namespace std;

const int TILE_WIDTH = 16;  

// 使用共享内存的矩阵乘法核函数
__global__ void MatrixMulSharedMemKernel(float *A, float *B, float *C, int wA, int wB) {
    // 块和线程的索引
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // 初始化Csub变量
    float Csub = 0;

    // 计算A和B的子矩阵
    for (int a = wA * TILE_WIDTH * by, b = TILE_WIDTH * bx; a < wA * TILE_WIDTH * by + wA - 1 && b < TILE_WIDTH * bx + wB - 1; a += TILE_WIDTH, b += TILE_WIDTH) {
        __shared__ float As[TILE_WIDTH][TILE_WIDTH + 1];  // 增加1以减少银行冲突
        __shared__ float Bs[TILE_WIDTH][TILE_WIDTH + 1];  // 增加1以减少银行冲突

        // 加载子矩阵A到共享内存
        int aRow = a / wA + ty;
        int aCol = a % wA + tx;
        if (aRow < wA && aCol < wA)
            As[ty][tx] = A[aRow * wA + aCol];
        else
            As[ty][tx] = 0.0f;

        // 加载子矩阵B到共享内存
        int bRow = b / wB + ty;
        int bCol = b % wB + tx;
        if (bRow < wA && bCol < wB)
            Bs[ty][tx] = B[bRow * wB + bCol];
        else
            Bs[ty][tx] = 0.0f;

        __syncthreads();  // 在加载完子矩阵后同步

        // 进行矩阵乘法
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();  // 确保计算完成后再加载下一个子矩阵
    }

    // 存储结果到矩阵C
    int row_C = by * TILE_WIDTH + ty;
    int col_C = bx * TILE_WIDTH + tx;
    if (row_C < wA && col_C < wB)
        C[row_C * wB + col_C] = Csub;
}

// CPU端的矩阵乘法计算
void matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB) {
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j) {
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}

// 比较CPU和GPU计算结果的差异
void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol) {
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i, j, k;
    int error_count = 0;
    for (j = 0; j < height; j++) {
        for (i = 0; i < width; i++) {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);
            if (fDiff > fListTol) {
                if (error_count < iListLength) {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }
                error_count++;
            }
        }
    }
    printf(" \n  Total Errors = %d\n", error_count);
}

// 解析命令行参数
void getArg(int argc, char *argv[], int &size, int &check) {
    if (argc != 3) {
        cerr << "Usage: " << argv[0] << " <check_enable> <size>\n";
        cerr << "\tcheck_enable: 1 to enable result checking\n";
        cerr << "\tsize: size of the matrix\n";
        exit(1);
    }
    int val1, val2;
    try {
        val1 = stoi(argv[1]);
        val2 = stoi(argv[2]);
    } catch (const invalid_argument &e) {
        cerr << "ERROR: parameters should be integer\n";
        exit(1);
    }
    check = val1;
    size = val2;
}

int main(int argc, char *argv[]) {
    int size, check;
    getArg(argc, argv, size, check);

    int m = size, n = size, k = size;

    // 声明存放在GPU上的数组
    float *h_M, *h_N, *d_M, *d_N;
    float *h_P, *d_P;
    size_t sizeM = m * k * sizeof(float);
    size_t sizeN = k * n * sizeof(float);
    size_t sizeP = m * n * sizeof(float);

    // Allocate host memory
    h_M = (float *)malloc(sizeM);
    h_N = (float *)malloc(sizeN);
    h_P = (float *)malloc(sizeP);
    float *reference = (float *)malloc(sizeP);

    // Allocate device memory
    hipMalloc(&d_M, sizeM);
    hipMalloc(&d_N, sizeN);
    hipMalloc(&d_P, sizeP);

    // Init data
    for (int i = 0; i < m * n; ++i) {
        if (i % 2 == 0)
            h_M[i] = 1.0;
        else
            h_M[i] = 0.5;
    }

    for (int i = 0; i < n * k; ++i) {
        if (i % 2 == 0)
            h_N[i] = 0.5;
        else
            h_N[i] = 1.0;
    }

    // Copy data from CPU to GPU
    hipMemcpy(d_M, h_M, sizeM, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, sizeN, hipMemcpyHostToDevice);

    // Timing records
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Launch kernel 定义grid&block
    dim3 grid((int)ceil(k * 1.0 / TILE_WIDTH), (int)ceil(m * 1.0 / TILE_WIDTH));
    dim3 block(TILE_WIDTH, TILE_WIDTH);

    int nIter = 5;
#ifdef USE_CUBLAS
    cublasHandle_t handle;
    cublasCreate(&handle);
#endif
    const float alpha = 1.0f;
    const float beta = 0.0f;
    for (int j = 0; j < nIter; j++) {
        // 你可以启用这个来使用CPU计算参考值
        // matrixMulCPU(reference, h_M, h_N, m, k, n);
        // 使用优化后的GPU矩阵乘法
        MatrixMulSharedMemKernel<<<grid, block>>>(d_M, d_N, d_P, m, n);
        // 使用cublasSgemm函数来计算矩阵乘法（可选）
        // cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, n, m, k, &alpha, d_N, n, d_M, k, &beta, d_P, n);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float msecPerMatrixMul;
    hipEventElapsedTime(&msecPerMatrixMul, start, stop);
    msecPerMatrixMul /= nIter;
    printf("Kernel Elapsed Time: %.3f ms\n", msecPerMatrixMul);

    // 计算和打印性能
    double flopsPerMatrixMul = 2.0 * (double)m * (double)n * (double)k;
    double gigaFlops =
        (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf("Performance= %.2f GFlop/s\n", gigaFlops);

    // Copy result back to host memory
    hipMemcpy(h_P, d_P, sizeP, hipMemcpyDeviceToHost);

    // Optionally check results
    if (check) {
        matrixMulCPU(reference, h_M, h_N, m, k, n);
        printDiff(reference, h_P, n, m, 10, 1e-5);
    }

    // Clean up
    free(h_M);
    free(h_N);
    free(h_P);
    free(reference);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return 0;
}
